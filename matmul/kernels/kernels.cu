#include <iostream>
#include "kernel1.cu"

template <typename KernelFunc>
float* runKernelAndGetResult(
    KernelFunc kernel,
    float *A, float *B, float *C, int M, int N, int K, float alpha, float beta, dim3 gridDim, dim3 blockDim){
    
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M*K*sizeof(float));
    hipMalloc((void**)&d_B, K*N*sizeof(float));
    hipMalloc((void**)&d_C, M*N*sizeof(float));
    
    hipMemcpy(d_A, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, M*N*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float* C_ref = new float[M*N];

    kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K, alpha, beta);
    hipMemcpy(C_ref, d_C, M*N*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K, alpha, beta);
    hipDeviceSynchronize();

    for(int i=0;i<10;i++){
        hipEventRecord(start);
        kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K, alpha, beta);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        std::cout << time << " ";
    }
    std::cout << "\n";
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return C_ref;
}